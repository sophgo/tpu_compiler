#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mlir/Dialect/TPU/GPUInplementation.h"

__global__ void ReLUForward(const int n, const float* in, float* out,
    float negative_slope) {
  CUDA_KERNEL_LOOP(i, n){
    out[i] = in[i] * ((in[i] > 0) + (in[i] <= 0) * negative_slope);
  }
}


int gpu_conv(float *input, float *weight, float *bias,
    float *output, int in, int ic, int ih, int iw, int oc, int oh, int ow,
    int kh, int kw, int sh, int sw, int dh, int dw, int ph, int pw, int g){
    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));
    // input
    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, in, ic, ih, iw));

    float *in_data;
    CUDA_CALL(hipMallocManaged(&in_data, in * ic * ih * iw * sizeof(float)));
    CUDA_CALL(hipMemcpy(in_data, input, in * ic * ih * iw * sizeof(float),
               hipMemcpyDefault));
    // filter
    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, oc, ic/g, kh, kw));


    float *filt_data;
    CUDA_CALL(hipMallocManaged(&filt_data, oc * ic * kh * kw / g * sizeof(float)));
    CUDA_CALL(hipMemcpy(filt_data, weight, oc * ic * kh * kw / g * sizeof(float),
               hipMemcpyDefault));

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc, ph, pw, sh, sw, dh,
                                                dw, HIPDNN_CROSS_CORRELATION,
                                                HIPDNN_DATA_FLOAT));
    // add group count(only on cudnn v7 or highter)
    hipdnnSetConvolutionGroupCount(conv_desc, g);

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT, in, oc, oh, ow));
    float *out_data;
    CUDA_CALL(hipMallocManaged(&out_data, in * oc * oh * ow * sizeof(float)));

    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
        cudnn, in_desc, filt_desc, conv_desc, out_desc,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));
    float *ws_data;
    CUDA_CALL(hipMallocManaged(&ws_data, ws_size));
    float alpha = 1, beta = 0;
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn, &alpha, in_desc, in_data, filt_desc, filt_data, conv_desc, algo,
        ws_data, ws_size, &beta, out_desc, out_data));

    if (bias) {
      beta = 1.0f;
      hipdnnTensorDescriptor_t bias_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT, 1, oc, 1, 1));

      float *biasdata;
      CUDA_CALL(hipMallocManaged(&biasdata, oc * sizeof(float)));
      CUDA_CALL(hipMemcpy(biasdata, bias, oc * sizeof(float),
                           hipMemcpyDefault));
      // add bias
      CUDNN_CALL(hipdnnAddTensor(cudnn, &alpha, bias_desc, biasdata, &beta,
                                out_desc, out_data));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(bias_desc));
      CUDA_CALL(hipFree(biasdata));
    }

    CUDA_CALL(hipMemcpy(output, out_data, in * oc * oh * ow * sizeof(float),
                         hipMemcpyDefault));
    hipDeviceSynchronize();
    // release
    CUDA_CALL(hipFree(ws_data));
    CUDA_CALL(hipFree(out_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDA_CALL(hipFree(filt_data));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDA_CALL(hipFree(in_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));
    return 0;
}

int gpu_relu(float *input, float *output, int n, int c, int h, int w,
             float negative_slope) {
  int size = n * c * h * w;
  float *gi, *go;
  hipMallocManaged(&gi, size * sizeof(float));
  hipMallocManaged(&go, size * sizeof(float));

  hipMemcpy(gi, input, size * sizeof(float), hipMemcpyDefault);
  ReLUForward<<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(size, gi, go,
                                                      negative_slope);
  hipDeviceSynchronize();
  hipMemcpy(output, go, size * sizeof(float), hipMemcpyDefault);
  hipFree(gi);
  hipFree(go);
  return 0;
}