#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mlir/Dialect/TPU/GPUInplementation.h"

__global__ void ReLUForward(const int n, const float* in, float* out,
    float negative_slope) {
  CUDA_KERNEL_LOOP(i, n){
    out[i] = in[i] * ((in[i] > 0) + (in[i] <= 0) * negative_slope);
  }
}

int gpu_relu(float *input, float *output, int n, int c, int h, int w,
             float negative_slope) {
  int size = n * c * h * w;
  float *gi, *go;
  hipMallocManaged(&gi, size * sizeof(float));
  hipMallocManaged(&go, size * sizeof(float));

  hipMemcpy(gi, input, size * sizeof(float), hipMemcpyDefault);
  ReLUForward<<<GET_BLOCKS(size), CUDA_NUM_THREADS>>>(size, gi, go,
                                                      negative_slope);
  hipDeviceSynchronize();
  hipMemcpy(output, go, size * sizeof(float), hipMemcpyDefault);
  hipFree(gi);
  hipFree(go);
  return 0;
}